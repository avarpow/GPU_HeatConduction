#include "hip/hip_runtime.h"
#include<iostream>
#include<string>
#include<cstring>
#include<chrono>

#include "mpi.h"
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
using namespace std;
#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)
float K;
float *data,*tmp_data;
float *device_data,*device_tmp_data;
int iterations;
int width,height;
chrono::time_point<chrono::system_clock> startTime;
float dataTransferTime=0;
int my_rank,my_size;
int grid_x;
int grid_y;
int x;
int y;
int x_start;
int x_end;
int y_start;
int y_end;
ncclUniqueId id;
ncclComm_t nccl_comm;
hipStream_t s;

__global__ void kernel(float *data,float *tmp_data,int x_start,int x_end,int y_start,int y_end,float K)
{
    int x=blockIdx.x*blockDim.x+threadIdx.x+x_start;
    int y=blockIdx.y*blockDim.y+threadIdx.y+y_start;
    if(x<width && y<height){
        if(x==0||x == width||y == 0||y == height){
            tmp_data[y*width+x]= data[y*width+x];
        }
        else{
            float delta = (K) * (data [(i - 1) * width + j] + data [(i + 1) * width + j] + data [i * width + j - 1] + data [i * width + j + 1] - 4 * data [i * width + j]);
            tmp_data [i * width + j] = data [i * width + j] + delta;
        }
    }
}
void draw_circle(float* data,int width,int height,float x,float y,float r,float val){
    for(int i=0;i<width;i++){
        for(int j=0;j<height;j++){
            float dx=i-x;
            float dy=j-y;
            if(dx*dx+dy*dy<r*r){
                data[i*height+j]=1;
            }
        }
    }
}
//prepare 数据准备,内存申请
void  prepare(){
    MPI_Comm_rank(MPI_COMM_WORLD,&my_rank);
    MPI_Comm_size(MPI_COMM_WORLD,&my_size);
    if (my_rank == 0) ncclGetUniqueId(&id);
    MPI_Bcast(&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);
    ncclCommInitRank(&comm, my_size, id, my_rank);
    if(my_size!=4){
        cout<<"error:only support 4 GPU"<<endl;
        exit(0);
    }
    
    grid_x = calGridSize(my_size);
    grid_y = my_size/grid_x;
    x = my_rank%grid_x;
    y = my_rank/grid_x;
    x_start = x*width/grid_x;
    x_end = (x+1)*width/grid_x;
    y_start = y*height/grid_y;
    y_end = (y+1)*height/grid_y;

    printf("my_rank:%d,grid_x:%d,grid_y:%d,x:%d,y:%d,x_start:%d,x_end:%d,y_start:%d,y_end:%d\n",my_rank,grid_x,grid_y,x,y,x_start,x_end,y_start,y_end);
    data =  (float*)malloc(sizeof(float)*width*height);
    tmp_data = (float*)malloc(sizeof(float)*width*height);
    memset(data,0,sizeof(float)*width*height);
    memset(tmp_data,0,sizeof(float)*width*height);
    if(my_rank==0){
        draw_circle(data,width,height,width/2,height/2,width/2,255);
    }
    CUDACHECK(hipSetDevice(localRank));
    CUDACHECK(hipStreamCreate(&s));

    hipMalloc(&device_data,sizeof(float)*width*height);
    hipMalloc(&device_tmp_data,sizeof(float)*width*height);
    hipMemcpy(device_data,data,sizeof(float)*width*height,hipMemcpyHostToDevice);
    // MPI_Bcast(data,width*height,MPI_FLOAT,0,MPI_COMM_WORLD);
    ncclBcast(device_data,width*height,ncclFloat,0,nccl_comm,s);
}
int calGridSize(int size){
    int min =999;
    int res =-1;
    for(int i=1;i<=size;i++){
        if(size%i==0){
            if((i+size/i)<min){
                min=(i+size/i);
                res=i;
            }
        }
    }
    return 0;
}

void prase_argv(int argc, char *argv[]){
    if(argc!=4){
        cout<<"Usage: "<<argv[0]<<" <K> <iterations> <width> <height>"<<endl;
        exit(1);
    }
    K=atof(argv[1]);
    iterations=atoi(argv[2]);
    width=atoi(argv[3]);
    height=atoi(argv[4]);
}
void finiaze(){
    hipMemcpy(data,device_data,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    hipFree(device_data);
    hipFree(device_tmp_data);
    free(data);
    free(tmp_data);

}
void multiGPUSolverReduce(){
    dim3 dimBlock(8,8);
    dim3 dimGrid(width/2/8,height/2/8);
    kernel<<<dimGrid,dimBlock>>>(device_data,device_tmp_data,x_start,x_end,y_start,y_end,K);
    auto time = chrono::system_clock::now();
    // MPI_Allreduce(tmp_data,data,width*height,MPI_FLOAT,MPI_SUM,MPI_COMM_WORLD);
    ncclAllReduce(device_tmp_data,device_data,width*height,ncclFloat,ncclSum,nccl_comm,NULL);
    dataTransferTime += endTimeCounter(time);
}
void startTimeCounter(chrono::time_point<chrono::system_clock> &startTime){
    startTime = chrono::system_clock::now();
}
float endTimeCounter(chrono::time_point<chrono::system_clock> &startTime){
    auto endTime = chrono::system_clock::now();
}

int main(int argc, char *argv[]) {
    prase_argv(argc, argv);
    prepare();
    startTimeCounter(startTime);
    multiGPUSolverReduce();
    endTimeCounter(startTime);
    finiaze();
    return 0;
}