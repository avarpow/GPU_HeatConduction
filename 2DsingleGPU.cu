#include "hip/hip_runtime.h"
#include<iostream>
#include<string>
#include<cstring>
#include<chrono>

// #include "mpi.h"
// #include "hip/hip_runtime.h"
// #include "hip/hip_runtime_api.h"
using namespace std;
float K;
float *data,*tmp_data;
float *device_data,*device_tmp_data;
int iterations;
int width,height;
chrono::time_point<chrono::system_clock> startTime;
__global__ void kernel(float *data,float *tmp_data,int width,int height)
{
    int x=blockIdx.x*blockDim.x+threadIdx.x;
    int y=blockIdx.y*blockDim.y+threadIdx.y;
    if(x<width||y<height){
        if (x == 0 || x == height - 1 || y == 0 || y == width - 1) {
            tmp_data [x * width + y] = data [x * width + y];
        }
        else {
            double delta = (K) * (data [(x - 1) * width + y] + data [(x + 1) * width + y] + data [x * width + y - 1] + data [x * width + y + 1] - 4 * data [x * width + y]);
            tmp_data [x * width + y] = data [x * width + y] + delta;
        }
    }
}
void draw_circle(float* data,int width,int height,float x,float y,float r,float val){
    for(int i=0;i<width;i++){
        for(int j=0;j<height;j++){
            float dx=i-x;
            float dy=j-y;
            if(dx*dx+dy*dy<r*r){
                data[i*height+j]=1;
            }
        }
    }
}
//prepare 数据准备,内存申请
void  prepare(){
    data =  (float*)malloc(sizeof(float)*width*height);
    tmp_data = (float*)malloc(sizeof(float)*width*height);
    hipMalloc(&device_data,sizeof(float)*width*height);
    hipMalloc(&device_tmp_data,sizeof(float)*width*height);
    memset(data,0,sizeof(float)*width*height);
    memset(tmp_data,0,sizeof(float)*width*height);
    draw_circle(data,width,height,width/2,height/2,width/2,255);
    hipMemcpy(device_data,data,sizeof(float)*width*height,hipMemcpyHostToDevice);
}
void singleGPUSolver(){
    dim3 dimBlock(8,8);
    dim3 dimGrid(width/8+1,height/8+1);
    kernel<<<dimGrid,dimBlock>>>(device_data,device_tmp_data,width,height);
    //更新新的数据
    hipMemcpy(device_data,device_tmp_data,sizeof(float)*width*height,hipMemcpyDeviceToDevice);
}
void finiaze(){
    hipMemcpy(data,device_data,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    free(data);
    free(tmp_data);
}
void prase_argv(int argc, char *argv[]){
    if(argc!=4){
        cout<<"Usage: "<<argv[0]<<" <K> <iterations> <width> <height>"<<endl;
        exit(1);
    }
    K=atof(argv[1]);
    iterations=atoi(argv[2]);
    width=atoi(argv[3]);
    height=atoi(argv[4]);
}
void startTimeCounter(chrono::time_point<chrono::system_clock> &startTime){
    startTime = chrono::system_clock::now();
}
void endTimeCounter(chrono::time_point<chrono::system_clock> &startTime){
    auto endTime = chrono::system_clock::now();
    printf("Time: %f\n", chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count() / 1000.0);
}
int main(int argc, char *argv[]) {
    prase_argv(argc, argv);
    prepare();
    startTimeCounter(startTime);
    singleGPUSolver();
    endTimeCounter(startTime);
    finiaze();
    return 0;
}