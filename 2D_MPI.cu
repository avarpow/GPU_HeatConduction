#include "hip/hip_runtime.h"
#include<iostream>
#include<string>
#include "EasyBMP.hpp"
#include "mpi.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
using namespace std;
using namespace EasyBMP;


class HeatConduction {
public:
    double K;
    double height;
    double width;
    double* data;
    double range_max = 1000;
    HeatConduction(double K, double height, double width) {
        this->K = K;
        this->height = height;
        this->width = width;
        data = new double [(int)height * (int)width];
    }
    void loadData(double* data) {
        for (int i = 0; i < height * width; i++) {
            this->data [i] = data [i];
        }
    }
    void loadDataFromFile(char* fileName) {
        FILE* file = fopen(fileName, "r");
        for (int i = 0; i < height * width; i++) {
            fscanf(file, "%lf", &data [i]);
        }
        fclose(file);
    }
    void saveDataToFile(char* fileName) {
        FILE* file = fopen(fileName, "w");
        for (int i = 0; i < height * width; i++) {
            fprintf(file, "%lf\n", data [i]);
        }
        fclose(file);
    }
    void MPIloadDataFromFile(char* fileName) {
        int rank;
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        if(rank == 0 ){
            FILE* file = fopen(fileName, "r");
            for (int i = 0; i < height * width; i++) {
                fscanf(file, "%lf", &data [i]);
            }
            fclose(file);
        }
        MPI_Bcast(data, height * width, MPI_DOUBLE, 0, MPI_COMM_WORLD);
    }
    void MPIsaveDataToFile(char* fileName) {
        int rank;
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        if(rank == 0 ){
            FILE* file = fopen(fileName, "w");
            for (int i = 0; i < height * width; i++) {
                fprintf(file, "%lf\n", data [i]);
            }
            fclose(file);
        }
    }
    void printData() {
        for (int i = 0; i < height * width; i++) {
            cout << data [i] << " ";
        }
        cout << endl;
    }
    double find_max() {
        double ret = 0;
        for (int i = 0; i < height; i++) {
            for (int j = 0; j < width; j++) {
                if (data [i * width + j] > ret) {
                    ret = data [i * width + j];
                }
            }
        }
        return ret;
    }
    void save2img(string filename) {
        double range_max = max(find_max(), this->range_max);
        if (height <= 1000 && width <= 1000) {
            EasyBMP::RGBColor black(0, 0, 0);  
            Image img(width, height,filename,black);
            for (int i = 0; i < height; i++) {
                for (int j = 0; j < width; j++) {
                    RGBColor color(data [i * width + j] / range_max, data [i * width + j] / range_max, data [i * width + j] / range_max);
                    color.r = 255-color.r;
                    color.g = 255-color.g;
                    color.b = 255-color.b;
                    img.SetPixel(j, i, color);
                }
            }
            img.Write();
        }
    }
    void MPIsave2img(string filename) {
        int rank;
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        if(rank == 0 ){
            double range_max = max(find_max(), this->range_max);
            if (height <= 1000 && width <= 1000) {
                EasyBMP::RGBColor black(0, 0, 0);  
                Image img(width, height,filename,black);
                for (int i = 0; i < height; i++) {
                    for (int j = 0; j < width; j++) {
                        RGBColor color(data [i * width + j] / range_max, data [i * width + j] / range_max, data [i * width + j] / range_max);
                        color.r = 255-color.r;
                        color.g = 255-color.g;
                        color.b = 255-color.b;
                        img.SetPixel(j, i, color);
                    }
                }
                img.Write();
            }
        }
    }
    //cpu单线程
    void cpuSolver(int iteration) {
        double* new_data = (double *)malloc(sizeof(double) * height * width);
        for(int k=0;k<iteration;k++){
            for(int i=0;i<height;i++){
                for(int j=0;j<width;j++){
                    //边界条件
                    if(i==0||i==height-1||j==0||j==width-1){
                        new_data[i*width+j]=data[i*width+j];
                    }
                    else{
                        double delta=(K)*(data[(i-1)*width+j]+data[(i+1)*width+j]+data[i*width+j-1]+data[i*width+j+1]-4*data[i*width+j]);
                        new_data[i*width+j]=data[i*width+j]+delta;
                    }
                }
            }
            //更新新的数据
            for(int i=0;i<height;i++){
                for(int j=0;j<width;j++){
                    data[i*width+j]=new_data[i*width+j];
                }
            }
        }
        free(new_data)
    }
    //单gpu
    /*
    block size: 16*16
    each thread : 16*16

    */
    const int row_per_thread = 16;
    const int col_per_thread = 16;
    void gpuSolver(int iteration) {
        dim3 dimBlock(16, 16);
        dim3 dimGrid((int)width / row_per_thread / dimBlock.x + 1 , (int)height / col_per_thread / dimBlock.y);
        double *d_data,*d_new_data;
        hipMalloc(&d_data, sizeof(double) * (int)height * (int)width);
        hipMalloc(&d_new_data, sizeof(double) * (int)height * (int)width);
        hipMemcpy(d_data, data, sizeof(double) * (int)height * (int)width, hipMemcpyHostToDevice);
        for(int k=0;k<iteration;k++){
            heatConductionKernel<<<dimGrid, dimBlock>>>(d_data, d_new_data, height, width, K);
        }
        hipMemcpy(data, d_new_data, sizeof(double) * (int)height * (int)width, hipMemcpyDeviceToHost);
        hipFree(d_data);
        hipFree(d_new_data);
    }
    //cpuMPI 实现
    void cpuMPIsolver(int iteration) {
        double* new_data = (double *)malloc(sizeof(double) * height * width);
        memset(new_data, 0, sizeof(double) * height * width);
        int rank, size;
        MPI_Comm_rank(MPI_COMM_WORLD, &rank);
        MPI_Comm_size(MPI_COMM_WORLD, &size);
        if(sqrt(size)*sqrt(size)!=size){
            cout<<"error: size must be a square number"<<endl;
            MPI_Finalize();
            exit(1);
        }
        int sqrt_size = sqrt(size);
        int row_per_process = (int)height / sqrt_size;
        int col_per_process = (int)width / sqrt_size;
        int row_start = rank / sqrt_size * row_per_process;
        int col_start = rank % sqrt_size * col_per_process;
        int row_end = row_start + row_per_process;
        int col_end = col_start + col_per_process;
        for(int k=0;k<iteration;k++){
            for(int i=row_start;i<row_end;i++){
                for(int j=col_start;j<col_end;j++){
                    //边界条件
                    if(i==0||i==height-1||j==0||j==width-1){
                        new_data[i*width+j]=data[i*width+j];
                    }
                    else{
                        double delta=(K)*(data[(i-1)*width+j]+data[(i+1)*width+j]+data[i*width+j-1]+data[i*width+j+1]-4*data[i*width+j]);
                        new_data[i*width+j]=data[i*width+j]+delta;
                    }
                }
            }
            MPI_Allreduce(new_data, new_data, height * width, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
            //更新新的数据
            for(int i=row_start;i<row_end;i++){
                for(int j=col_start;j<col_end;j++){
                    data[i*width+j]=new_data[i*width+j];
                }
            }
        }
    }
    //gpuMPI 实现
    
    void gpuMPIsolver(int iteration) {
        const int GPU_PER_NODE = 4;
        //TODO
        
    }

};
int main() {
    MPI_Init(NULL, NULL);

    MPI_Finalize();
    return 0;
}